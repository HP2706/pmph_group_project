#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdint>
#include "tests/test_transpose_ker.cu"
#include "tests/test_scan_inc.cu"

int main() {
    initHwd();

    // setup params

    const uint32_t input_size = 1000000;
    const uint32_t Q = 22;
    const uint32_t lgH = 8;
    const uint32_t BLOCK_SIZE = 256;
    const uint32_t GRID_SIZE = 1024;
    const uint32_t T = 6;
    const uint32_t ELEMS_PER_THREAD_SCAN = 32;
    const uint32_t grid_size = (input_size + (BLOCK_SIZE * Q - 1)) / (BLOCK_SIZE * Q);
    using P = Params<uint32_t, uint32_t, Q, lgH, GRID_SIZE, BLOCK_SIZE, T, ELEMS_PER_THREAD_SCAN>;

    test_verify_transpose<P>(input_size);


}
