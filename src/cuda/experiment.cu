#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include "constants.cuh"
#include <iostream>
#include <cstdint>
#include "tests/test_transpose_ker.cu"
#include "tests/test_radix_sort_ker.cu"
#include "tests/test_histo_ker.cu"
#include "tests/test_glb_to_reg.cu"
#include "tests/test_two_way_partition.cu"
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <vector>





int main() {
    initHwd();

    // setup params

    const uint32_t input_size = 10000;
    const uint32_t Q = 22; // 22
    const uint32_t lgH = 8;
    const uint32_t BLOCK_SIZE = 256;
    const uint32_t TILE_SIZE = 32;


    
    const uint32_t NUM_BLOCKS = (input_size + (BLOCK_SIZE * Q - 1)) / (BLOCK_SIZE * Q);

    printf("total number of threads used: %u\n", NUM_BLOCKS * BLOCK_SIZE);

    using P = Params<
        uint32_t, 
        uint32_t, 
        Q, 
        lgH, 
        NUM_BLOCKS, 
        BLOCK_SIZE, 
        TILE_SIZE
    >;

    test_verify_transpose<P>(input_size);
    
    
    //TestTwoWayPartition<P>();
    //test_call_rank_permute_ker<P>(input_size);
    //test_glb_to_reg_ker<P>(input_size);
    //test_count_sort<P>(input_size);
    //printf("CountSort done\n");
    //test_histo_ker<P>(input_size);
    //test_radix_sort_ker<P>(input_size);

}
