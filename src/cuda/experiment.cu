#include "kernels.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include "helper.h"
#include "helper_kernels/rank_permute.cuh"




int main() {

    uint32_t* h_in;
    uint32_t* d_in;
    uint32_t* d_out;
    uint32_t* d_histogram;
    uint32_t* h_histogram;
    
    uint32_t* d_histogram_transposed;
    uint32_t* d_hist_out;
    uint32_t* h_histogram_transposed;
    

    const uint32_t SIZE = 1000000;
    const uint32_t NUM_BINS = 1 << 8;
    const uint32_t BLOCK_SIZE = 1024;
    const uint32_t Q = 22;
    const uint32_t lgH = 8;
    
    // Calculate grid size based on input size and elements per thread
    const uint32_t grid_size = (SIZE + (BLOCK_SIZE * Q - 1)) / (BLOCK_SIZE * Q);
    // Change the histogram size calculation
    const uint32_t hist_size = NUM_BINS * grid_size; // This needs to be calculated before PrepareMemory


    PrepareMemory<uint32_t, BLOCK_SIZE>(
        &h_in, 
        &d_in, 
        &d_histogram, 
        &h_histogram,
        NUM_BINS,
        SIZE,
        hist_size
    );

    uint32_t* h_hist_out = (uint32_t*) malloc(sizeof(uint32_t) * hist_size);
    // initialize h_histogram_transposed to 0
    h_histogram_transposed = (uint32_t*) malloc(sizeof(uint32_t) * hist_size);
    for (int i = 0; i < hist_size; i++) {
        h_histogram_transposed[i] = 0;
        h_hist_out[i] = 0;
    }

    hipMalloc((uint32_t**) &d_hist_out, sizeof(uint32_t) * hist_size);
    hipMemcpy(d_hist_out, h_hist_out, sizeof(uint32_t) * hist_size, hipMemcpyHostToDevice);
    hipMemset(d_hist_out, 0, sizeof(uint32_t) * hist_size);


    hipMalloc((uint32_t**) &d_histogram_transposed, sizeof(uint32_t) * hist_size);
    hipMemcpy(d_histogram_transposed, h_histogram_transposed, sizeof(uint32_t) * hist_size, hipMemcpyHostToDevice);
    hipMemset(d_histogram_transposed, 0, sizeof(uint32_t) * hist_size);

    // initialize d_out to 0
    hipMalloc((uint32_t**) &d_out, sizeof(uint32_t) * SIZE);
    hipMemset(d_out, 0, sizeof(uint32_t) * SIZE);

    uint32_t* h_out = (uint32_t*) malloc(sizeof(uint32_t) * SIZE);

    uint32_t height = NUM_BINS;
    uint32_t width = grid_size;

    randomInit<uint32_t>(h_histogram, hist_size, NUM_BINS);
    hipMemcpy(d_histogram, h_histogram, sizeof(uint32_t) * hist_size, hipMemcpyHostToDevice);

    printf("SIZE: %d, height: %d, width: %d, height times width: %d\n", hist_size, height, width, height * width);

    printf("height: %d, width: %d\n", height, width);
    // Update the kernel call with correct dimensions

    using SortParams = Params<uint32_t, uint32_t, Q, lgH, grid_size, BLOCK_SIZE, 32>;
    CountSort<SortParams>(
        d_in,
        d_out,
        d_histogram,
        d_histogram_transposed,
        d_hist_out,
        SIZE,
        uint32_t(0)
    );


    

    // Add after kernel call
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
    }



    hipMemcpy(h_histogram, d_histogram, sizeof(uint32_t) * hist_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_histogram_transposed, d_histogram_transposed, sizeof(uint32_t) * hist_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_hist_out, d_hist_out, sizeof(uint32_t) * hist_size, hipMemcpyDeviceToHost);

    printf("h_histogram: ");
    for (int i = 0; i < hist_size; i++) {
        if (0 == h_histogram[i]) {
            printf("%d ", h_histogram[i]);
        }
    }
    printf("\n");

    
    printf("\n");


    return 0;

}
