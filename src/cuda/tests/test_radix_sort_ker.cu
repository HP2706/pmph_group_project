#include "hip/hip_runtime.h"
#pragma once

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>
#include <math.h>
#include <cstdint>

#include "../constants.cuh"
#include "../helper.h"
#include "../helper_kernels/utils.cuh"
#include "../helper_kernels/pbb_kernels.cuh"
#include "../helper_kernels/prefix_sum.cuh"
#include "../kernels.cuh"
#include "../cub_kernel.cuh"
#include <hip/hip_runtime.h>

template<typename P>
__host__ void test_radix_sort_ker(
    uint32_t input_size
)
{
    static_assert(is_params<P>::value, "P must be a Params instance");

    // declare ptrs
    typename P::ElementType* cub_h_in;
    typename P::ElementType* cub_d_in;
    typename P::ElementType* cub_h_out;
    typename P::ElementType* cub_d_out;
    
    typename P::ElementType* d_in;
    typename P::ElementType* d_out;
    typename P::ElementType* h_out;


    allocateAndInitialize<typename P::ElementType, P::MAXNUMERIC_ElementType>(
        &h_out,
        &d_out,
        input_size,
        false
    );

    // allocate memory
    allocateAndInitialize<typename P::ElementType, P::MAXNUMERIC_ElementType>(
        &cub_h_in,
        &cub_d_in,
        input_size,
        true
    );

    allocateAndInitialize<typename P::ElementType, P::MAXNUMERIC_ElementType>(
        &cub_h_out,
        &cub_d_out,
        input_size,
        false
    );


    hipMalloc((typename P::ElementType**) &d_in, input_size * sizeof(typename P::ElementType));
    hipMemcpy(d_in, cub_d_in, input_size * sizeof(typename P::ElementType), hipMemcpyDeviceToDevice);




    CUBSortKernel<
        typename P::ElementType, 
        P::BLOCK_SIZE, 
        P::Q
    ><<<P::GRID_SIZE, P::BLOCK_SIZE>>>
    (
        cub_d_in, 
        cub_d_out, 
        input_size
    );

    // copy results back to host
    hipMemcpy(cub_h_out, cub_d_out, input_size * sizeof(typename P::ElementType), hipMemcpyDeviceToHost);


    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("cub sort kernel failed: %s\n", hipGetErrorString(err));
        return;
    }


    RadixSortKer<P>(
        d_in,
        d_out,
        input_size
    );


    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("radix sort kernel failed: %s\n", hipGetErrorString(err));
        return;
    }

    // Add debug prints for input data
    printf("\nFirst 20 input elements:\n");
    for (uint32_t i = 0; i < 20 && i < input_size; i++) {
        printf("cub_h_in[%d]: %d\n", i, cub_h_in[i]);
    }

    hipMemcpy(h_out, d_out, input_size * sizeof(typename P::ElementType), hipMemcpyDeviceToHost);

    // Find and print the exact location where sorting fails
    for (uint32_t i = 1; i < input_size; i++) {
        if (h_out[i] < h_out[i-1]) {
            printf("\nRadixSort failure at index %d: %d > %d\n", 
                i, h_out[i-1], h_out[i]);
            // Print surrounding elements for context
            printf("Elements around failure point:\n");
            for (int j = max(0, (int)i-5); j < min(input_size, i+5); j++) {
                printf("h_out[%d]: %d\n", j, h_out[j]);
            }
            break;
        }
    }

    if (!checkSorted(h_out, input_size)) {
        printf("RadixSortKer is not sorted\n");
        // print top 100 elements
        printf("top 100 elements:\n");
        for (uint32_t i = 0; i < 100; i++) {
            printf("h_out[%d]: %d\n", i, h_out[i]);
        }
    } else {
        printf("RadixSortKer is sorted\n");
    }

    // Find and print the exact location where CUBSort fails
    for (uint32_t i = 1; i < input_size; i++) {
        if (cub_h_out[i] < cub_h_out[i-1]) {
            printf("\nCUBSort failure at index %d: %d > %d\n", 
                i, cub_h_out[i-1], cub_h_out[i]);
            // Print surrounding elements for context
            printf("Elements around failure point:\n");
            for (int j = max(0, (int)i-5); j < min(input_size, i+5); j++) {
                printf("cub_h_out[%d]: %d\n", j, cub_h_out[j]);
            }
            break;
        }
    }

    if (!checkSorted(cub_h_out, input_size)) {
        printf("CUBSortKernel is not sorted\n");
        // print top 100 elements
        printf("top 100 elements:\n");
        for (uint32_t i = 0; i < 100; i++) {
            printf("cub_h_out[%d]: %d\n", i, cub_h_out[i]);
        }
    } else {
        printf("CUBSortKernel is sorted\n");
    }

    printf("checking cub matches radix\n");
    assert(validate(cub_h_out, h_out, input_size));

}