#include "hip/hip_runtime.h"
#pragma once

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>
#include <math.h>
#include <cstdint>
#include <vector>

#include "../constants.cuh"
#include "../helper.h"
#include "../helper_kernels/utils.cuh"
#include "../helper_kernels/rank_permute.cuh"
#include "../kernels.cuh"
#include "../cub_kernel.cuh"
#include <hip/hip_runtime.h>



template<class P>
__global__ void testGlbToRegKernel(
    typename P::ElementType* arr_in,
    typename P::ElementType* arr_out,
    uint64_t N                       // Total number of elements
) {
    uint32_t tid = threadIdx.x;
    uint32_t bid = blockIdx.x;
    using uint = typename P::ElementType;

    extern __shared__ uint64_t sh_mem_uint64[];
    uint* shmem = (uint*) sh_mem_uint64;

    uint reg[P::Q];

    GlbToReg<uint, P::Q, P::BLOCK_SIZE, P::MAXNUMERIC_ElementType>(N, shmem, arr_in, reg);

    __syncthreads();
    
    const uint32_t QB = P::BLOCK_SIZE * P::Q;
    const uint64_t glb_offs = bid * QB;
    
    for (int i = 0; i < P::Q; i++) 
    {
        uint64_t expected_idx = glb_offs + tid * P::Q + i;
        uint expected_value = (expected_idx < N) ? arr_in[expected_idx] : P::MAXNUMERIC_ElementType;

        //Assert or print mismatches if any
        assert(reg[i] == expected_value);  // Optionally, use printf for non-assert debugging
    }
    __syncthreads();
}

template<typename P>
bool verifyGlbToReg(
    const typename P::ElementType* arr_in, 
    const typename P::ElementType* arr_out, 
    uint64_t N,
    int grid_size
) 
{
    bool success = true;
    const uint32_t QB = P::BLOCK_SIZE * P::Q;

    for (uint32_t bid = 0; bid < grid_size; ++bid) 
    {
        for (uint32_t tid = 0; tid < P::BLOCK_SIZE; ++tid) 
        {
            const uint32_t thread_start = bid * QB + tid * P::Q;
            for (int i = 0; i < P::Q; i++) 
            {
                uint64_t idx = thread_start + i;
                typename P::ElementType expected_value = (idx < N) ? arr_in[idx] : P::MAXNUMERIC_ElementType;

                if (arr_out[idx] != expected_value) 
                {
                    printf("Mismatch at index %d: expected %d, got %d\n", idx, expected_value, arr_out[idx]);
                    success = false;
                }
            }
        }
    }
    return success;
}

using ElementType = uint32_t;

template<typename P>
__host__ void test_glb_to_reg_ker(
    uint32_t N,
    int grid_size
)
{
    typename P::ElementType* h_in;
    typename P::ElementType* d_in;
    
    typename P::ElementType* h_out;
    typename P::ElementType* d_out;

    allocateAndInitialize<typename P::ElementType, P::MAXNUMERIC_ElementType>(
        &h_in, 
        &d_in, 
        N,
        true
    );

    allocateAndInitialize<typename P::ElementType, P::MAXNUMERIC_ElementType>(
        &h_out, 
        &d_out, 
        N,
        false
    );

    //Set shared memory size (assuming it fits the necessary elements)
    size_t shared_mem_size = P::BLOCK_SIZE * P::Q * sizeof(ElementType);

    //Launch kernel
    testGlbToRegKernel<P><<<grid_size, P::BLOCK_SIZE, P::QB*sizeof(typename P::ElementType)>>> 
    (
        d_in,
        d_out,
        N,
        grid_size
    );
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
    {
        printf("Test kernel failed: %s\n", hipGetErrorString(err));
        return;
    }
    else
    {
        printf("GlbToRegKernel passed!\n");
    }

    //Free device memory
    hipFree(d_in);
    hipFree(d_out);
}
