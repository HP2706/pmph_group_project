#pragma once

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>
#include <math.h>
#include <cstdint>

#include "../constants.cuh"
#include "../helper.h"
#include "../helper_kernels/utils.cuh"
#include "../helper_kernels/pbb_kernels.cuh"
#include "../helper_kernels/prefix_sum.cuh"
#include "../kernels.cuh"
#include "../cub_kernel.cuh"
#include <hip/hip_runtime.h>

template<typename P>
__host__ void test_count_sort(
    uint32_t input_size
)
{
    static_assert(is_params<P>::value, "P must be a Params instance");

    // for this test we use uint8_t and thus count sort should fully sort the input
    // assuming lgH is 8 

    assert(P::lgH == 8);
    assert(sizeof(typename P::ElementType) * 8 == P::lgH && "Element type bits must match lgH ");

    uint32_t hist_size = P::H * P::GRID_SIZE;

    // ptr allocations
    typename P::ElementType* d_in;
    typename P::ElementType* h_in;

    typename P::ElementType* d_out;
    typename P::ElementType* h_out;

    typename P::UintType* d_hist;
    typename P::UintType* d_hist_transposed;
    typename P::UintType* d_hist_scanned;
    typename P::UintType* d_tmp;

    // cub allocations
    typename P::ElementType* cub_d_in;
    typename P::ElementType* cub_d_out;
    typename P::ElementType* cub_h_out;

    allocateAndInitialize<typename P::ElementType>(
        &h_in,
        &d_in,
        input_size,
        true // we initialize to random values
    );

    allocateAndInitialize<typename P::ElementType>(
        &h_out,
        &d_out,
        input_size,
        false // we initialize to 0
    );

    allocateAndInitialize<typename P::UintType>(
        nullptr,
        &d_hist,
        hist_size,
        false // we initialize to 0
    );

    allocateAndInitialize<typename P::UintType>(
        nullptr,
        &d_hist_transposed,
        hist_size,
        false // we initialize to 0
    );

    allocateAndInitialize<typename P::UintType>(
        nullptr,
        &d_hist_scanned,
        hist_size,
        false // we initialize to 0
    );

    allocateAndInitialize<typename P::UintType>(
        nullptr,
        &d_tmp,
        hist_size,
        false // we initialize to 0
    );

    CountSort<P>(
        d_in, 
        d_out, 
        d_hist, 
        d_hist_transposed, 
        d_hist_scanned, 
        d_tmp, 
        input_size, 
        0 // we use a bit position of 0 for this test
    );

    hipMemcpy(h_out, d_out, sizeof(typename P::ElementType) * input_size, hipMemcpyDeviceToHost);

    {
        // we free some memory
        hipFree(d_out);
        hipFree(d_hist);
        hipFree(d_hist_transposed);
        hipFree(d_hist_scanned);
        hipFree(d_tmp);
    }


    // cub allocations
    hipMalloc((void**)&cub_d_in, sizeof(typename P::ElementType) * input_size);
    // we keep an array of the input in host memory to compare the results
    hipMemcpy(cub_d_in, h_in, sizeof(typename P::ElementType) * input_size, hipMemcpyHostToDevice);

    // we check cub_d_in == d_in
    // this is just a dummy check
    assert(validate(cub_d_in, d_in, input_size));
    
    
    allocateAndInitialize<typename P::ElementType>(
        &cub_h_out,
        &cub_d_out,
        input_size,
        false 
    );
    
    CubKer<
        typename P::ElementType, 
        P::GRID_SIZE, 
        P::BLOCK_SIZE, 
        P::Q
    >(
        cub_d_in,
        cub_d_out,
        input_size
    );

    /* 
    CUBSortKernel<
        typename P::ElementType, 
        P::BLOCK_SIZE, 
        P::Q
    ><<<P::GRID_SIZE, P::BLOCK_SIZE>>>
    (
        cub_d_in,
        cub_d_out,
        input_size
    ); */

    hipMemcpy(cub_h_out, cub_d_out, sizeof(typename P::ElementType) * input_size, hipMemcpyDeviceToHost);
    assert(validate(cub_h_out, h_out, input_size));

    {
        // we free some gpu-memory
        hipFree(cub_d_in);
        hipFree(cub_d_out);

        // we free some host-memory
        free(cub_h_out);
        free(h_in);
        free(h_out);
    }
   
}


